#include "hip/hip_runtime.h"
//cpu
void squarecpu(unsigned*matrix,unsigned *result,unsigned matrixsize /* 64*/){
    for(unsigned ii = 0;ii < matrixsize;ii++){
        for(unsigned jj = 0;jj < matrixsize;jj++){
            for(unsigned kk = 0;kk<matrixsize;kk++){
                result[ii*matrixsize+jj]=result[ii*matrixsize+kk]*matrix[kk*matrixsize+jj];
            }
        }
    }

}

//gpu version 1
square<<<1,N>>>(matrix,result,N);
__global__ void square(unsigned*matrix,unsigned *result,unsigned matrixsize){
        unsigned id = blockIdx.x*blockDim.x + threadIdx.x;
        for(unsigned jj = 0;jj < matrixsize;jj++){
            for(unsigned kk = 0;kk<matrixsize;kk++){
                result[id * matrixsize+jj]+=
                matrix[id * matrixsize +kk]*
                matrix[kk * matrixsize +jj];
            }
        }
}

//gpu version 2
square<<<N,N>>>(matrix,result,N);
__global__ void square(unsigned*matrix,unsigned *result,unsigned matrixsize){
        unsigned id = blockIdx.x*blockDim.x + threadIdx.x;
        unsigned ii = id / matrixsize;
        unsigned jj = id % matrixsize;
        for(unsigned kk = 0;kk < matrixsize;kk++){
            result[ii * matrixsize+jj]+=
            matrix[ii * matrixsize +kk]*
            matrix[kk * matrixsize +jj];
        }
        
}