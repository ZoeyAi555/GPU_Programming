#include "hip/hip_runtime.h"
__global__ void dkernel(unsigned*vector,unsigned vectorsize){
    unsigned id = blockIdx.x*blockDim.x+threadId.x;
    if(id%2) vector[id] = id;
    else vector[id] = vectorsize*vectorsize;
    vector[id]++;
}
/*
Wrap: Differnt wrap-thread execute different instructions, thraed are said to diverge
Diverge
Degree of Divergence; # steps to complete one instruction for each thread in the wrap
*/

//rewrite the following program to remove thread-divergence: 
int x,y,z;
if(x==y) x=z; 
else x=y;

int mask = (x == y); // 1 if x equals y, 0 otherwise
int value_if_true = z;
int value_if_false = y;

// Perform both operations for all threads
int result_if_true = value_if_true;
int result_if_false = value_if_false;

// Combine the results based on the mask
x = mask * result_if_true + (!mask) * result_if_false;
